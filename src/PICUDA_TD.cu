#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cstring>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "PBM1d.cpp"
#include "PICUDA.cu"

#define BLOCK_SIZE 16
#define RAIO 1

using namespace std;

void chamaKernelTD(PBM1d* in, PBM1d* out, int N, size_t sizeMat);
__global__ void vet_td_gpu_g_kernel(int* IN, int* OUT);

/// TRANSFORMADA DA DISTANCIA em 2D
PBM1d* PICUDA::TD2D(PBM1d* img) {
  int i;
  int HEIGTH = img->getHeight();
  int WIDTH = img->getWidth();
  int N = img->getTam();

  PBM1d* in = new PBM1d();
  in->copyOf(img);
  PBM1d* out = new PBM1d();
  out->zerado( HEIGTH, WIDTH);

  int ALTO = HEIGTH * WIDTH;
  for(i = 0; i < ALTO; i++) {
    in->set1(i, in->get1(i)*ALTO);
  }

  // tamanho de bytes da img (ela eh mod 16)
  size_t sizeMat = (sizeof (int) * in->getTam() );

  /// chama Horizontalmente
  chamaKernelTD(in, out, N, sizeMat);

  // copia OUT -> IN
  memcpy(in->getMat(), out->getMat(), sizeMat);

  // faz a transposta da img
  in->transpose();

  /// chama Verticalmente
  chamaKernelTD(in, out, N, sizeMat);

  // faz a transposta da img
  out->transpose();

  delete in;
  return out;
}

void chamaKernelTD(PBM1d* in, PBM1d* out, int N, size_t sizeMat) {
  int ite = 0;
  while(1) {
    // Aloca o IN e o OUT na GPU
    int* d_IN;
    int* d_OUT;
    hipMalloc((void**)&d_IN, sizeMat);
    hipMalloc((void**)&d_OUT, sizeMat);

    // Copia o IN da CPU para a GPU
    hipMemcpy(d_IN, (void*)in->getMat(), sizeMat, hipMemcpyHostToDevice);

    // Lanca Kernel
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid( N / dimBlock.x, 1);
    vet_td_gpu_g_kernel <<< N / BLOCK_SIZE, BLOCK_SIZE >>> (d_IN + RAIO, d_OUT + RAIO);

    // Copia o OUT da GPU para a CPU
    hipMemcpy((void*)out->getMat(), d_OUT, sizeMat, hipMemcpyDeviceToHost);

    // Desaloca da GPU
    hipFree(d_IN);
    hipFree(d_OUT);

    // verifica se nao houve mudanca ( se IN==OUT entao PARA)
    if( PBM1d::imgIGUAL(in, out) ) break;

    // copia OUT -> IN
    memcpy(in->getMat(), out->getMat(), sizeMat);

    if(ite > in->getWidth() + 1) {
      break;
    }
  }
}

__global__ void vet_td_gpu_g_kernel(int* IN, int* OUT) {
  __shared__ int temp[BLOCK_SIZE + 2 * RAIO];
  int Gid = threadIdx.x + blockIdx.x * blockDim.x;
  int Lid = threadIdx.x + RAIO;

  // Adiciona os elementos na memoria Shared
  temp[Lid] = IN[Gid];
  if (threadIdx.x < RAIO) {
    temp[Lid - RAIO] = IN[Gid - RAIO];
    temp[Lid + BLOCK_SIZE] = IN[Gid + BLOCK_SIZE];
  }

  // Sincroniza pra garantir q todos so dados vao estar disponivel
  __syncthreads();

  // aplica
  int menor = INT_MAX;
  if(temp[Lid] == 0) {
    // se o analizado eh 0
    OUT[Gid] = 0;
  } else {
    // escolhe o menor de seus 2 vizinhos
    for (int vizinho = -RAIO ; vizinho <= RAIO ; vizinho++)
      menor = min(menor, temp[Lid + vizinho]);

    // Salva o menorado + 1
    if(menor >= temp[Lid] ) { // (sem estourar o limite, no cazo de dimensao grande)
      OUT[Gid] = temp[Lid];
    } else {
      OUT[Gid] = menor + 1;
    }
  }
}
