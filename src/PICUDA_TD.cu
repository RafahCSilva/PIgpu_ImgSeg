#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cstring>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "PBM1d.cpp"
#include "PICUDA.cu"

#define BLOCK_SIZE 16
#define RAIO 1

using namespace std;

void chamaKernelTD_global(PBM1d* in, PBM1d* out, int N, size_t sizeMat);
__global__ void vet_td_gpu_g_kernel(int* IN, int* OUT);
void chamaKernelTD_shared(PBM1d* in, PBM1d* out, int N, size_t sizeMat);
__global__ void vet_td_gpu_s_kernel(int* IN, int* OUT);


void PICUDA::TD2D_multGPU(PBM1d* img) {
  PBM1d* out;

  // Nas duas gpu
  int devicesCount, deviceIndex;
  hipGetDeviceCount(&devicesCount);
  for( deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, deviceIndex);
    hipSetDevice(deviceIndex);
    printf("%s \n", deviceProperties.name );

    // GPU global memory
    TEMPO_tic();
    PBM1d* imgG = PICUDA::TD2D_GLOBAL(img1);
    TEMPO_toc_TD();
    cout << " ";
    delete imgG;

    // GPU shared memory
    TEMPO_tic();
    PBM1d* imgS = PICUDA::TD2D_SHARED(img1);
    TEMPO_toc_TD();
    cout << endl;
    delete imgS;
  }

}


/// =============================== GLOBAL MEMORY

/// TRANSFORMADA DA DISTANCIA em 2D com GLOBAL MEMORY
PBM1d* PICUDA::TD2D_GLOBAL(PBM1d* img) {
  int i;
  int HEIGTH = img->getHeight();
  int WIDTH = img->getWidth();
  int N = img->getTam();

  PBM1d* in = new PBM1d();
  in->copyOf(img);
  PBM1d* out = new PBM1d();
  out->zerado( HEIGTH, WIDTH);

  int ALTO = HEIGTH * WIDTH;
  for(i = 0; i < ALTO; i++) {
    in->set1(i, in->get1(i)*ALTO);
  }

  // tamanho de bytes da img (ela eh mod 16)
  size_t sizeMat = (sizeof (int) * in->getTam() );

  /// chama Horizontalmente
  chamaKernelTD_global(in, out, N, sizeMat);

  // copia OUT -> IN
  memcpy(in->getMat(), out->getMat(), sizeMat);

  // faz a transposta da img
  in->transpose();

  /// chama Verticalmente
  chamaKernelTD_global(in, out, N, sizeMat);

  // faz a transposta da img
  out->transpose();

  delete in;
  return out;
}

void chamaKernelTD_global(PBM1d* in, PBM1d* out, int N, size_t sizeMat) {
  int ite = 0;
  while(1) {
    // Aloca o IN e o OUT na GPU
    int* d_IN;
    int* d_OUT;
    hipMalloc((void**)&d_IN, sizeMat);
    hipMalloc((void**)&d_OUT, sizeMat);

    // Copia o IN da CPU para a GPU
    hipMemcpy(d_IN, (void*)in->getMat(), sizeMat, hipMemcpyHostToDevice);

    // Lanca Kernel
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid( N / dimBlock.x, 1);
    vet_td_gpu_g_kernel <<< N / BLOCK_SIZE, BLOCK_SIZE >>> (d_IN + RAIO, d_OUT + RAIO);

    // Copia o OUT da GPU para a CPU
    hipMemcpy((void*)out->getMat(), d_OUT, sizeMat, hipMemcpyDeviceToHost);

    // Desaloca da GPU
    hipFree(d_IN);
    hipFree(d_OUT);

    // verifica se nao houve mudanca ( se IN==OUT entao PARA)
    if( PBM1d::imgIGUAL(in, out) ) break;

    // copia OUT -> IN
    memcpy(in->getMat(), out->getMat(), sizeMat);

    if(ite > in->getWidth() + 1) {
      break;
    }
  }
}

__global__ void vet_td_gpu_g_kernel(int* IN, int* OUT) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;

  // aplica
  int menor = INT_MAX;
  if(IN[id] == 0) {
    // se o analizado eh 0
    OUT[id] = 0;
  } else {
    // escolhe o menor de seus 2 vizinhos
    for (int vizinho = -RAIO ; vizinho <= RAIO ; vizinho++)
      menor = min(menor, IN[id + vizinho]);

    // Salva o menor + 1
    if(menor >= IN[Lid] ) { // (sem estourar o limite, no cazo de dimensao grande)
      OUT[id] = IN[Lid];
    } else {
      OUT[id] = menor + 1;
    }
  }
}


/// =============================== SHARED MEMORY

/// TRANSFORMADA DA DISTANCIA em 2D com SHARED MEMORY
PBM1d* PICUDA::TD2D_SHARED(PBM1d* img) {
  int i;
  int HEIGTH = img->getHeight();
  int WIDTH = img->getWidth();
  int N = img->getTam();

  PBM1d* in = new PBM1d();
  in->copyOf(img);
  PBM1d* out = new PBM1d();
  out->zerado( HEIGTH, WIDTH);

  int ALTO = HEIGTH * WIDTH;
  for(i = 0; i < ALTO; i++) {
    in->set1(i, in->get1(i)*ALTO);
  }

  // tamanho de bytes da img (ela eh mod 16)
  size_t sizeMat = (sizeof (int) * in->getTam() );

  /// chama Horizontalmente
  chamaKernelTD_shared(in, out, N, sizeMat);

  // copia OUT -> IN
  memcpy(in->getMat(), out->getMat(), sizeMat);

  // faz a transposta da img
  in->transpose();

  /// chama Verticalmente
  chamaKernelTD_shared(in, out, N, sizeMat);

  // faz a transposta da img
  out->transpose();

  delete in;
  return out;
}

void chamaKernelTD_shared(PBM1d* in, PBM1d* out, int N, size_t sizeMat) {
  int ite = 0;
  while(1) {
    // Aloca o IN e o OUT na GPU
    int* d_IN;
    int* d_OUT;
    hipMalloc((void**)&d_IN, sizeMat);
    hipMalloc((void**)&d_OUT, sizeMat);

    // Copia o IN da CPU para a GPU
    hipMemcpy(d_IN, (void*)in->getMat(), sizeMat, hipMemcpyHostToDevice);

    // Lanca Kernel
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid( N / dimBlock.x, 1);
    vet_td_gpu_s_kernel <<< N / BLOCK_SIZE, BLOCK_SIZE >>> (d_IN + RAIO, d_OUT + RAIO);

    // Copia o OUT da GPU para a CPU
    hipMemcpy((void*)out->getMat(), d_OUT, sizeMat, hipMemcpyDeviceToHost);

    // Desaloca da GPU
    hipFree(d_IN);
    hipFree(d_OUT);

    // verifica se nao houve mudanca ( se IN==OUT entao PARA)
    if( PBM1d::imgIGUAL(in, out) ) break;

    // copia OUT -> IN
    memcpy(in->getMat(), out->getMat(), sizeMat);

    if(ite > in->getWidth() + 1) {
      break;
    }
  }
}

__global__ void vet_td_gpu_s_kernel(int* IN, int* OUT) {
  __shared__ int temp[BLOCK_SIZE + 2 * RAIO];
  int Gid = threadIdx.x + blockIdx.x * blockDim.x;
  int Lid = threadIdx.x + RAIO;

  // Adiciona os elementos na memoria Shared
  temp[Lid] = IN[Gid];
  if (threadIdx.x < RAIO) {
    temp[Lid - RAIO] = IN[Gid - RAIO];
    temp[Lid + BLOCK_SIZE] = IN[Gid + BLOCK_SIZE];
  }

  // Sincroniza pra garantir q todos so dados vao estar disponivel
  __syncthreads();

  // aplica
  int menor = INT_MAX;
  if(temp[Lid] == 0) {
    // se o analizado eh 0
    OUT[Gid] = 0;
  } else {
    // escolhe o menor de seus 2 vizinhos
    for (int vizinho = -RAIO ; vizinho <= RAIO ; vizinho++)
      menor = min(menor, temp[Lid + vizinho]);

    // Salva o menor + 1
    if(menor >= temp[Lid] ) { // (sem estourar o limite, no cazo de dimensao grande)
      OUT[Gid] = temp[Lid];
    } else {
      OUT[Gid] = menor + 1;
    }
  }
}
