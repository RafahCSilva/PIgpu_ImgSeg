#include "hip/hip_runtime.h"
/*
  Processamento de Imagens na GPU
    Rafael Cardoso da Silva    21048012
      Segmentacao de Imagem com o Algoritmo de Watershed
      Versao em CPU
 */
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <cstring>
#include "PBM1d.cpp"
#include "PICUDA.cu"
#include "TEMPO.cpp"

using namespace std;

int main(int argc, char const *argv[]) {

  if (argc < 3) {
    std::cerr << "Usage: " << argv[0] << " <ORIGEM> <DESTINO> [-v, -vv, -bench]" << std::endl;
    return 1;
  }

  const char * entrada = argv[1];
  const char * saida = argv[2];

  bool verbose1 = false;
  bool verbose2 = false;
  bool bench = false;

  if (argc == 4) {
    if( string(argv[3]) == "-v") {
      verbose1 = true;
      verbose2 = false;
    } else if( string(argv[3]) == "-vv") {
      verbose1 = true;
      verbose2 = true;
    } else if( string(argv[3]) == "-bench") {
      verbose1 = false;
      verbose2 = false;
      bench = true;
    } else {
      cerr << "Terceito paramentro invalido!" << std::endl;
      std::cerr << "Usage: " << argv[0] << " <ORIGEM> <DESTINO> [-v, -vv, -bench]" << std::endl;
      exit(1);
    }
  }

  if(verbose1) {
    cout << "Processamento de Imagens na GPU" << endl;
    cout << "  Rafael Cardoso da Silva    21048012" << endl;
    cout << "    Segmentacao de Imagem com o Algoritmo de Watershed" << endl;
    cout << "    Versao em CPU" << endl;
  }

  double tempo = 0;

  if(verbose1)    cout << "\nCarregando Imagem de Entrada... ";
  PBM1d* img1 = new PBM1d();
  img1->loadAsP1(entrada);
  if(verbose1)    cout << "OK\n";
  if(verbose2)    img1->print();

  if(verbose1)    cout << "\nTD\n";
  if(verbose1)    TEMPO_tic();
  if(bench)       TEMPO_tic();
  PBM1d* img2 = PICUDA::TD2D(img1);
  if(bench)       tempo += TEMPO_toc_bench();
  if(verbose1)    TEMPO_toc();
  if(verbose2)    img2->print();

  img2->saveAsP1_("TD2.pbm");

  if(verbose1)    cout << "\nCORTE ( abaixo de ";
  int corte = ceil( img2->MAX() );
  if(verbose1)    cout << corte << " )\n";
  if(verbose1)    TEMPO_tic();
  if(bench)       TEMPO_tic();
  PBM1d* img3 = PICUDA::CORTE(img2, corte);
  if(bench)       tempo += TEMPO_toc_bench();
  if(verbose1)    TEMPO_toc();
  if(verbose2)    img3->print();

  if(verbose1)    cout << "\nBINARIO\n";
  if(verbose1)    TEMPO_tic();
  if(bench)       TEMPO_tic();
  PBM1d* img4 = PICUDA::BINARIO(img3);
  if(bench)       tempo += TEMPO_toc_bench();
  if(verbose1)    TEMPO_toc();
  if(verbose2)    img4->print();

  if(verbose1)    cout << "\nLABEL\n";
  if(verbose1)    TEMPO_tic();
  if(bench)       TEMPO_tic();
  PBM1d* img5 = PICUDA::LABEL(img4);
  if(bench)       tempo += TEMPO_toc_bench();
  if(verbose1)    TEMPO_toc();
  if(verbose2)    img5->print();

  if(verbose1)    cout << "\nWATERSHED\n";
  if(verbose1)    TEMPO_tic();
  if(bench)       TEMPO_tic();
  PBM1d* img6 = PICUDA::WATERSHED(img5, img1);
  if(bench)       tempo += TEMPO_toc_bench();
  if(verbose1)    TEMPO_toc();
  if(verbose2)    img6->print();

  if(verbose1)    cout << "\nDESTACANDO BORDA\n";
  if(verbose1)    TEMPO_tic();
  if(bench)       TEMPO_tic();
  PBM1d* img7 = PICUDA::GETBORDA(img6);
  if(bench)       tempo += TEMPO_toc_bench();
  if(verbose1)    TEMPO_toc();
  if(verbose2)    img7->print();

  if(verbose1)    cout << "\nSalvando Imagem de Saida... ";
  img7->saveAsP1(saida);
  if(verbose1)    cout << "OK\n";

  if(bench) cout << tempo << endl;

  delete img1;
  delete img2;
  delete img3;
  delete img4;
  delete img5;
  delete img6;
  delete img7;

  return 0;
}
